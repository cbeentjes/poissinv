#include "hip/hip_runtime.h"
//
// standard header files
//

#include <stdio.h>
#include <stdlib.h>

//
// CUDA header file
//

#include <hip/hip_runtime.h>

//
// my header file
//

#include "poissinv_cuda.h"

// number of samples per thread
#define SAMPLES 100.0

__global__ void normcdfinvf_test(int M) {

    float x, u;
    int   tid = threadIdx.x + blockIdx.x*blockDim.x;

    u = (tid + 0.5f) / M;

    for (int n=0; n<SAMPLES; n++) {
        u += 1e-10f;

        x = normcdfinvf(u);

        // needed to prevent compiler discarding everything
        if (x==-999.0f) printf("negative x\n");
    }
}

__global__ void poissinvf_test(int M, float lam) {

    float x, u;
    int   tid = threadIdx.x + blockIdx.x*blockDim.x;

    u = (tid + 0.5f) / M;

    for (int n=0; n<SAMPLES; n++) {
        u += 1e-10f;

        // Mixed case used when lam < 0
        // lam takes values from {1,2,4,8,16,32,64,128}
        if (lam<0.0f) {
            int n = 1 << (tid & 7);
            lam   = 1.0f * (float) n;
        }
        x = poissinvf(u, lam);

        // needed to prevent compiler discarding everything
        if (x==-999.0f) printf("negative x\n");
    }
}

__global__ void normcdfinv_test(int M) {

    float x, u;
    int   tid = threadIdx.x + blockIdx.x*blockDim.x;

    u = (tid + 0.5f) / M;

    for (int n=0; n<SAMPLES; n++) {
        u += 1e-10f;

        x = normcdfinv((double) u);

        // needed to prevent compiler discarding everything
        if (x==-999.0f) printf("negative x\n");
    }
}

__global__ void poissinv_test(int M, float lam) {

    float x, u;
    int   tid = threadIdx.x + blockIdx.x*blockDim.x;

    u = (tid + 0.5f) / M;

    for (int n=0; n<SAMPLES; n++) {
        u += 1e-10f;

        // Mixed case used when lam < 0
        // lam takes values from {1,2,4,8,16,32,64,128}
        if (lam<0.0f) {
            int n = 1 << (tid & 7);
            lam   = 1.0f * (float) n;
        }
        x = poissinv((double) u, (double) lam);

        // needed to prevent compiler discarding everything
        if (x==-999.0f) printf("negative x\n");
    }
}



//
// main code
//

int main(int argc, char **argv) {
    float lam;
    int   M, nblocks, nthreads;

    // number of loop iterations in timing tests
#ifndef COUNT_LAMBDA
#define COUNT_LAMBDA 16
#endif

    // CUDA timing

    float milli;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // set number of blocks, and threads per block

    M = (1<<27);
    nthreads = 256;
    nblocks  = M / nthreads;

    // execute kernels

    for (int pass=0; pass<2; pass++) {
        // Option to test only double prec or single prec codes
#ifdef DOUBLE
        if (pass % 2 == 0) continue;
#endif
#ifdef SINGLE
        if (pass % 2 == 1) continue;
#endif

        if (pass==0)
            printf("\nsingle precision performance tests (GPU)\n");
        else
            printf("\ndouble precision performance tests (GPU)\n");
        printf("---------------------------------- \n");
        printf("  lambda   execution time   samples/sec \n");

        // default parameter values
        lam = 0.125f;

        // change lambda based on input if desired
        if (argc > 1) {
            lam = strtod(argv[1], NULL);
        }

        /* Fixed parameters */
        for (int count=0; count<=COUNT_LAMBDA; count++) {

            hipEventRecord(start);

            if (pass==0)
                poissinvf_test<<<nblocks,nthreads>>>(M, lam);
            else
                poissinv_test<<<nblocks,nthreads>>>(M, lam);

            hipEventRecord(stop);
            hipEventSynchronize(stop);
            hipEventElapsedTime(&milli, start, stop);

            // factor SAMPLES due to repeat in test routines
            // factor 1e3 due to timing in milliseconds
            if (count>0) { // skip first one for more accurate timing (cache effects?)
                printf("   %6g      %9.4f     %10.3g \n",
                        lam, milli, float(M)*SAMPLES*1e3/milli);
            }
#ifdef PLUS
            lam += 1.0f;
#else
            lam *= 2.0f;
#endif
        }

        /* Mixed parameters */
        // Run once first for more accurate timing
        if (pass == 0)
            poissinvf_test<<<nblocks,nthreads>>>(M, -1.0f);
        else
            poissinv_test<<<nblocks,nthreads>>>(M, -1.0f);

        hipEventRecord(start);
            if (pass == 0)
                poissinvf_test<<<nblocks,nthreads>>>(M, -1.0f);
            else
                poissinv_test<<<nblocks,nthreads>>>(M, -1.0f);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milli, start, stop);
        printf("    mixed      %9.4f     %10.3g \n",
                milli, float(M)*SAMPLES*1e3/milli);

        /* normcdfinv */
        if (pass == 0)
            normcdfinvf_test<<<nblocks,nthreads>>>(M);
        else
            normcdfinv_test<<<nblocks,nthreads>>>(M);

        hipEventRecord(start);
        if (pass == 0)
            normcdfinvf_test<<<nblocks,nthreads>>>(M);
        else
            normcdfinv_test<<<nblocks,nthreads>>>(M);

        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milli, start, stop);

        printf("\n normcdfinv    %9.4f     %10.3g \n",
                milli, float(M)*SAMPLES*1e3/milli);

    }

    // CUDA exit -- needed to flush printf write buffer

    hipDeviceReset();
    return 0;
}
